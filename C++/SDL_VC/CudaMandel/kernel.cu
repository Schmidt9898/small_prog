#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "Mandelgpu.h"
#include <iostream>

#define Ddim(arr,x,y) arr[(y) + (x)*YY]

__device__ struct komplex
{
    double i, v;
    __device__ komplex(double vv, double ii)
    {
        i = ii;
        v = vv;
    }
    __device__ double re()
    {
        return(v);
    }
    __device__ double im()
    {
        return (i);
    }
};
__device__ komplex  operator+ (komplex a, komplex b)
{
    a.i += b.i;
    a.v += b.v;
    return a;
}
__device__ komplex  operator* (komplex a, komplex b)
{
    komplex c(0, 0);
    c.i = a.v * b.i + a.i * b.v;
    c.v = a.i * b.i * -1 + (a.v * b.v);
    return c;
}


__global__ void cudamandelkernel(float* data, const int XX, const int YY,const int MAX, const double mag, const double xoff, const double yoff) {
    
        int x = threadIdx.x + blockIdx.x * blockDim.x;
        int y = threadIdx.y + blockIdx.y * blockDim.y;
        if (x < XX && y < YY) {

            double cr = (mag / double(XX)) * x - mag / 2 - xoff;
            double ci = (mag / double(YY)) * y - mag / 2 - yoff;
                komplex c(cr, ci);
                komplex z(0, 0);
                float t = 0;
                while (z.re() * z.re() + z.im() * z.im() < 4.0 && t < MAX) {
                    z = z * z + c;
                    t+=1;
                }
                //gout << move_to(x, y) << color(t, t, t) << dot;

                Ddim(data, x, y) = t/MAX*256;
        }
    

}
float* mandelgpuinit(const int XX, const int YY)
{
    float* datavram;
    hipMalloc(&datavram, XX*YY*sizeof(int));
    return datavram;
}

void mandelGPU(float* __restrict datavram,float* __restrict dataram, const int XX, const int YY,const int MAX, const double mag, const double xoff, const double yoff) {

    dim3 threads(32,16,1);
    dim3 blocks((XX - 1) / 32 + 1, (YY - 1) / 16 + 1, 1);
    cudamandelkernel<<<blocks, threads >>> (datavram, XX,YY,MAX,mag,xoff,yoff);
    hipMemcpy(dataram, datavram, XX*YY*sizeof(int), hipMemcpyDeviceToHost);
}