#include "hip/hip_runtime.h"
﻿#include "myheader.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

__global__ void f() {
    int x = threadIdx.x, y=threadIdx.y, z=threadIdx.z;
    printf(" Hello from gpu #%d , %d , %d\n",x,y,z);
}
void F()
{
    dim3 threads(10);

    f <<<1, threads >> > ();
    hipDeviceSynchronize();
    printf("hello from cpu\n");
}
/*
int main()
{
    dim3 threads(10);

    f <<<1,threads>>> ();
    hipDeviceSynchronize();
    printf("hello from cpu\n");


    return 0;
}
*/
