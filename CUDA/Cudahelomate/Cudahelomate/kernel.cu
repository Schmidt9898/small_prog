#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>


__global__ void helokernel()
{
    int i = threadIdx.x;
    printf("Helo mate from gpu!\n");
}

int main()
{
    printf("Helo this is the host cpu\n");
    dim3 threads(10);
    helokernel << <1, threads >> > ();
    hipDeviceSynchronize();
    system("pause");
    return 0;

}